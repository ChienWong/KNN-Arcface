#include "hip/hip_runtime.h"


#include <hip/device_functions.h>


#include<hip/device_functions.h>
#include<math_functions.h>

#include <vector>

#define  PI 3.1415926
/**
@brief ����
*/
struct Matrix
{
	int width;
	int height;
	float *elements;
};
/**
@brief ��ȡ����Ԫ��
@param �������
@param Ԫ��������
@param Ԫ��������
@return ����Ԫ��ֵ
*/
__device__ float getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}
/**
@brief ���þ���
@param �������
@param Ԫ��������
@param Ԫ��������
@param Ԫ��ֵ
*/
__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}
/**
@brief ����*����
@note �÷����������ھ����������block��������rows,threadpreblock��������cols
@param �������
@param ��������
@param ��������
*/
__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	__shared__ float shared[512];
	shared[tid] = getElement(A, bid, tid) * getElement(B, tid, 0);
	for (int stride = 2; stride <= 512; stride *= 2) {
		if (tid%stride == 0) {
			shared[tid] += shared[tid + stride / 2];
		}
		__syncthreads();
	}
	if (tid == 0) {
		shared[0] = acos(shared[0]) / PI * 180;
		setElement(C, bid, 0, shared[0]);
	}
}
/**
@brief ���㵱ǰ�����������Ѵ��������ľ���
@param �Ѵ洢����
@param ��ǰ����
@return ���о���
*/
std::vector<float> getAllangle(std::vector<const float*> set,const float* _f) {
	Matrix *A, *B, *C;
	// �����й��ڴ�
	hipMallocManaged((void**)&A, sizeof(Matrix));
	hipMallocManaged((void**)&B, sizeof(Matrix));
	hipMallocManaged((void**)&C, sizeof(Matrix));
	A->height = set.size();
	A->width = 512;
	B->height = 512;
	B->width = 1;
	C->width = 1;
	C->height = set.size();
	hipMallocManaged((void**)&A->elements, A->width * A->height * sizeof(float));
	hipMallocManaged((void**)&B->elements, B->width * B->height * sizeof(float));
	hipMallocManaged((void**)&C->elements, C->width * C->height * sizeof(float));
	for (int i = 0; i < set.size(); i++) {
		for (int j = 0; j < 512; j++) {
			A->elements[i * 512 + j] = set[i][j];
		}
	}
	for (int i = 0; i < 512; i++) {
		B->elements[i] = _f[i];
	}
	dim3 blockSize(512);
	dim3 gridSize(A->height);
	matMulKernel <<< gridSize, blockSize >>>(A, B, C);
	const hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess)\
	{
		printf("ERROR: %s:%d,", __FILE__, __LINE__);
		printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
	}
	std::vector<float> temp;
	for (int i = 0; i < C->height; i++) {
		printf("the element %d : %f\n", i, C->elements[i]);
		temp.push_back(C->elements[i]);
	}
	hipFree(A->elements);
	hipFree(B->elements);
	hipFree(C->elements);
	hipFree(A);
	hipFree(B);
	hipFree(C);
	return temp;
}